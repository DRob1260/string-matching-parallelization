#include "hip/hip_runtime.h"
#ifndef __INCLUDES_H_
#define __INCLUDES_H_
#include "naive_search_cuda.cuh"
#include "search_utils.h"
#include <stdlib.h>
#include <stdbool.h>
#include <stdio.h>
#endif

__global__ void cudaFunction(char *pattern, int patternSize, char *target, long targetSize, int numThreads, SearchResult *searchResults[]) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    long searchSize = targetSize / numThreads;
    long start = threadIndex * searchSize;
    long end = start + searchSize;
    if(threadIndex+1 == numThreads) {
        end = targetSize-1;
    }

    SearchResult searchResult;
    searchResult.matchIndexes = (long *)malloc((searchSize / patternSize) * sizeof(long));
    searchResult.matchTotal = 0;

    for(long t=start; t < end; t++) {
        bool isMatch = true;
        for(int p=0; p < patternSize; p++) {
            if(target[t+p] != pattern[p]) {
                isMatch = false;
                break;
            }
        }

        if(isMatch) {
            searchResult.matchIndexes[searchResult.matchTotal] = t;
            searchResult.matchTotal++;
        }
    }

    *searchResults[threadIndex] = searchResult;
}

SearchResult naiveSearchParallelCuda(char *pattern, int patternSize, char *target, long targetSize, int numThreads, int blockSize) {
    printf("Starting Naive Parallel Search using Cuda.\n");
    printf("patternSize: %i\n", patternSize);
    printf("targetSize: %li\n", targetSize);

    double startTime = getTime();

    SearchResult searchResult;
    searchResult.searchType = "Naive Parallel Cuda Search";
    searchResult.pattern = pattern;
    searchResult.matchIndexes = (long *)malloc((targetSize / patternSize) * sizeof(long));
    searchResult.matchTotal = 0;
   
    SearchResult *searchResults[numThreads]; 
    int numBlocks = ceil(double(numThreads)/blockSize);
    cudaFunction<<<numBlocks, blockSize>>>(pattern, patternSize, target, targetSize, numThreads, searchResults);
    hipDeviceSynchronize();

    int currentMatchIndex = 0;
    for(int i=0; i < numThreads; i++) {
        SearchResult *sr = searchResults[i];
        searchResult.matchTotal += sr->matchTotal;
        for(int x=0; x < sr->matchTotal; x++) {
            searchResult.matchIndexes[currentMatchIndex] = sr->matchIndexes[x];
            currentMatchIndex++;
        }
    }

    searchResult.duration = getTime() - startTime;

    return searchResult;
}
