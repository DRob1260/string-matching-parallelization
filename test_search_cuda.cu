#ifndef __INCLUDES_H_
#define __INCLUDES_H_
#include "naive_search_cuda.cuh"
#include "search_utils.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#endif

// todo: malloc needs to be replaced with hipMallocManaged
int main(int argc, char* argv[]) {

    if(argc < 7) {
        printf("Invalid arguments. Usage: alignments_filepath genome_filepath pattern_length number_of_patterns number_of_threads block_size target_length_limit(optional)\n");
        exit(1);
    }
    int patternLength = atoi(argv[3]);
    int numPatterns = atoi(argv[4]);
    int numThreads = atoi(argv[5]);
    int blockSize = atoi(argv[6]);
    long targetLengthLimit = -1;
    char *alignmentsFilepath = malloc(100 * sizeof(char));
    char *genomeFilePath = malloc(100 * sizeof(char));
    strncpy(alignmentsFilepath, argv[1], 100);
    strncpy(genomeFilePath, argv[2], 100);
    printf("Input alignments filepath: %s\n", alignmentsFilepath);
    printf("Input genome filepath: %s\n", genomeFilePath);
    printf("Input pattern length: %i\n", patternLength);
    printf("Input number of patterns: %i\n", numPatterns);
    printf("Input number of threads: %i\n", numThreads);
    if(argc == 8) {
        targetLengthLimit = atol(argv[7]);
        printf("Input target length limit: %li\n", targetLengthLimit);
    }

    char *patterns[numPatterns];
    buildSearchPatterns(patterns, alignmentsFilepath, patternLength, numPatterns);

    SearchTarget searchTarget = buildSearchTarget(genomeFilePath, targetLengthLimit);
    printf("SearchTarget length: %li\n", searchTarget.targetLength);

    SearchResult searchResult;

    // filename format is naive_parallel_cuda_<numThreads>_threads.csv
    char filename[100];
    strcat(filename, "naive_parallel_cuda_");
    char numThreadsStr[16];
    sprintf(numThreadsStr, "_%i_threads.csv", numThreads);
    strcat(filename,numThreadsStr);

    FILE *searchResultsFile = initializeSearchResultsFile(filename);
    for(int i=0; i < numPatterns; i++) {
        searchResult = naiveSearchParallelCuda(patterns[i], patternLength, searchTarget.target, searchTarget.targetLength, numThreads, blockSize);
        printSearchResults(searchResult);
        writeSearchResultToFile(searchResultsFile, searchResult);
    }
}